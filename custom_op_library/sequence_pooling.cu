#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation. All rights reserved.
// Licensed under the MIT License.

#include <hip/hip_fp16.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdint>
#include "sequence_pooling.h"

using namespace std;

// An example
// In: input: [1, 4096, 768]
// In: sen_lens: [1, 47]     contains like [30, 40, 20, ....., 96] and sum up to 4096
// Out: output: [1, 256, 768]
//      where [0, 0:46, 768] is the max pooling result of input along axis=1 by sen_lens
//      and [0, 47:256, 768] part is all zeros


template <typename InputIt, typename OutputIt>
__device__ inline void PrefixSumLinear(const InputIt* first, const InputIt* last, OutputIt* d_first) {
  if (first == last) {
    return;
  }
  OutputIt sum = *first;
  *d_first = sum;
  while (++first != last) {
    sum += *first;
    *(++d_first) = sum;
  }
}

template <typename T>
__global__ void SequencePoolingCudaKernel(const T* input, const int64_t* sentence_lengthes, const int num_sequences, const int sequence_length_for_split, T* output) {

  const int hidden_size = gridDim.z;
  const int num_sequences_max = blockDim.x;
  const int batch_id = blockIdx.x;
  const int hidden_id = blockIdx.z;
  const int seq_id_per_batch = threadIdx.x;

  int sentence_lengthes_prefixsum[256]; // num_sequences <= 256

  const int offset = batch_id * num_sequences;
  const int num_sequences_limit = num_sequences < 256 ? num_sequences : 256;

  PrefixSumLinear(sentence_lengthes + offset, sentence_lengthes + offset + num_sequences_limit, sentence_lengthes_prefixsum);

  const int past_sequence_length = (seq_id_per_batch == 0) ? 0 : sentence_lengthes_prefixsum[seq_id_per_batch - 1];

  const int input_offset = batch_id * hidden_size * sequence_length_for_split + hidden_size * past_sequence_length + hidden_id;
  const int output_offset = batch_id * hidden_size * num_sequences_max + hidden_size * seq_id_per_batch + hidden_id;

  if (sentence_lengthes[seq_id_per_batch + offset + 1] == 0) {
    output[output_offset] = 0;
  } else if (seq_id_per_batch == 255) {
    output[output_offset] = 0;
  } else {
    T local_max = (T)0;
    const int sequence_length = sentence_lengthes[seq_id_per_batch + offset];

    for (int i = 0; i < sequence_length; ++i) {
      if (i == 0) {
        local_max = *(input + input_offset);
      } else {
        T value = *(input + input_offset + i * hidden_size);
        local_max = (float)value > (float)local_max ? value : local_max;
      }
    }
    output[output_offset] = local_max;
  }

}

void SequencePoolingCuda(
  hipStream_t stream,
  const int batch_size,
  const int hidden_size,
  const int num_sequences,
  const int sequence_length_for_split,
  const float* input,
  const int64_t* sentence_lengthes,
  float* output) 
{
  const int num_sequences_max = 256;
  const dim3 grid(batch_size, 1, hidden_size);
  const dim3 block(num_sequences_max, 1, 1);
  
  SequencePoolingCudaKernel<float><<<grid, block, 0, stream>>>(input, sentence_lengthes, num_sequences, sequence_length_for_split, output);
}

void SequencePoolingCuda(
  hipStream_t stream,
  const int batch_size,
  const int hidden_size,
  const int num_sequences,
  const int sequence_length_for_split,
  const half* input,
  const int64_t* sentence_lengthes,
  half* output) 
{
  const int num_sequences_max = 256;
  const dim3 grid(batch_size, 1, hidden_size);
  const dim3 block(num_sequences_max, 1, 1);

  SequencePoolingCudaKernel<half><<<grid, block, 0, stream>>>(input, sentence_lengthes, num_sequences, sequence_length_for_split, output);
}


